#include "hip/hip_runtime.h"
﻿#include "hough.cuh"
#include <fstream>
#include <hip/hip_math_constants.h>
using namespace std;

__global__ void accumulate_edge_points(uint8_t* image, int image_size, uint8_t* edges, unsigned int* edges_len) {
	//Setup shared memory variables
	extern __shared__ uint8_t sh_mem[];
	unsigned int* sh_next = (unsigned int*)sh_mem;
	unsigned int* sh_global_write = (unsigned int*)(sh_mem + 4);
	uint8_t* sh_edges = sh_mem + 8;

	//Initialize shared memory
	if (threadIdx.x == 0) {
		*sh_next = 0;
	}
	__syncthreads();

	int image_x = threadIdx.x;
	int image_y = blockIdx.x;

	//Bring in the proper pixel from global memory
	int pixel = image[image_y * image_size + image_x];
	//If the pixel is part of an edge
	if (pixel == 1) {
		unsigned int write_ind = atomicAdd(sh_next, (unsigned int)2);
		//Write the point to shared memory
		sh_edges[write_ind] = image_x + 1;
		sh_edges[write_ind + 1] = image_y + 1;
	}
	__syncthreads();

	//Figure out where we need to start writing our portion in global memory
	if (threadIdx.x == 0) {
		*sh_global_write = atomicAdd(edges_len, *sh_next);
	}
	__syncthreads();

	//Write our shared memory to global memory
	if (threadIdx.x < *sh_next) {
		edges[*sh_global_write + threadIdx.x] = sh_edges[threadIdx.x];
	}
}

__global__ void hough(uint8_t* edges, unsigned int* edges_len, int* global_acc) {
	//(Image_size/shrink)^2 * 3 different radius sizes
	//(256/4) = 64
	__shared__ unsigned int hough[64 * 64 * 3];

	//for (int i = 0; i < 64; i++) {
	//	for (int j = 0; j < 3; j++) {
	//		hough[threadIdx.x * 64 + i * 64 + j];
	//	}
	//}

	for (int i = threadIdx.x; i < 64 * 64 * 3; i += blockDim.x) {
		hough[i] = 0;
	}

	__syncthreads();

	for (int k = threadIdx.x; k < (*edges_len) / 2; k += blockDim.x) {
		uint8_t point_x = edges[k * 2];
		uint8_t point_y = edges[k * 2 + 1];

		float shrunk_y = (float)point_y / 4;
		float shrunk_x = (float)point_x / 4;

		/*if (k == 0) {
			printf("POINT: %d,%d\n", point_x, point_y);
		}*/

		for (int i = 1; i < 361; i++) {
			float sin_result = sinf((i * HIP_PI_F) / 180);
			float cos_result = cosf((i * HIP_PI_F) / 180);

			for (int j = 1; j <= 3; j++) {
				int a = round(shrunk_x - (5 + j) * sin_result);
				int b = round(shrunk_y - (5 + j) * cos_result);
				/*if (k == 0 && j == 1) {
					printf("%d,%d\n", a, b);
				}*/
				/*if (a == 6 && b == 0 && j == 1) {
					printf("%d", 1);
				}*/
				if (0 <= a && a < 64 && 0 <= b && b < 64) {
					atomicAdd(&hough[a + b * 64 + 64 * 64 * (j - 1)], (unsigned int)1);
					//hough[a + b * 64 + 64 * 64 * (j - 1)] += 1;
				}
			}
		}
	}

	__syncthreads();

	for (int i = threadIdx.x; i < 64 * 64 * 3; i += blockDim.x) {
		global_acc[i] = hough[i];
	}
}