﻿#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string>
#include "hip/hip_runtime.h"
#include ""

#include "hough.cuh"
using namespace std;

#define IMAGE_SIZE 256
#define NUM_IMAGES 100
#define ACCUMULATOR_SIZE 64
#define NUM_RADIUS 3
#define TOTAL_ACC_SIZE ACCUMULATOR_SIZE * ACCUMULATOR_SIZE * NUM_RADIUS

int start()
{
	const string input_dir = "C:\\Users\\djkong7\\Documents\\GitHub\\CS759_SP21_FinalProject\\processed_images\\edges\\binary\\";
	const string output_dir = "C:\\Users\\djkong7\\Documents\\GitHub\\CS759_SP21_FinalProject\\processed_images\\hough\\binary\\";
	uint8_t* image = new uint8_t[IMAGE_SIZE * IMAGE_SIZE];
	float accumulate_time = 0.0;
	float hough_time = 0.0;

	hipError_t cuda_stat;
	uint8_t* dev_image;
	int* dev_edges_x;
	int* dev_edges_y;
	int* dev_edges_len;
	int* acc = new int[TOTAL_ACC_SIZE];
	int* dev_acc;

	// allocate image memory on the device(GPU)
	cuda_stat = hipMalloc((void**)&dev_image, sizeof(uint8_t) * IMAGE_SIZE * IMAGE_SIZE);
	if (cuda_stat != hipSuccess) {
		printf("device image memory allocation failed");
		return EXIT_FAILURE;
	}

	// allocate edges x memory on the device(GPU)
	cuda_stat = hipMalloc((void**)&dev_edges_x, sizeof(int) * IMAGE_SIZE * IMAGE_SIZE);
	if (cuda_stat != hipSuccess) {
		printf("device edges x memory allocation failed");
		return EXIT_FAILURE;
	}

	// allocate edges y memory on the device(GPU)
	cuda_stat = hipMalloc((void**)&dev_edges_y, sizeof(int) * IMAGE_SIZE * IMAGE_SIZE);
	if (cuda_stat != hipSuccess) {
		printf("device edges y memory allocation failed");
		return EXIT_FAILURE;
	}

	// allocate edges length memory on the device(GPU)
	cuda_stat = hipMalloc((void**)&dev_edges_len, sizeof(int));
	if (cuda_stat != hipSuccess) {
		printf("device edges length memory allocation failed");
		return EXIT_FAILURE;
	}

	// allocate accumulator memory on the device(GPU)
	cuda_stat = hipMalloc((void**)&dev_acc, sizeof(int) * TOTAL_ACC_SIZE);
	if (cuda_stat != hipSuccess) {
		printf("device accumulator memory allocation failed");
		return EXIT_FAILURE;
	}

	for (int z = 1; z <= NUM_IMAGES; z++) {
		ifstream my_file(input_dir + "image-" + to_string(z), ios::in | ios::binary);
		if (my_file.is_open()) {
			my_file.read((char*)image, IMAGE_SIZE * IMAGE_SIZE);
			my_file.close();
		}
		else {
			cout << "File not opened";
			return 0;
		}

		// put the image on the device
		cuda_stat = hipMemcpy(dev_image, image, sizeof(uint8_t) * IMAGE_SIZE * IMAGE_SIZE, hipMemcpyHostToDevice);
		if (cuda_stat != hipSuccess) {
			printf("image move to device failed");
			return EXIT_FAILURE;
		}

		// Initialize the global points length to 0
		cuda_stat = hipMemset((void*)dev_edges_len, 0, sizeof(int));
		if (cuda_stat != hipSuccess) {
			printf("device edges length memset failed");
			return EXIT_FAILURE;
		}

		// Start timer code
		hipEvent_t start;
		hipEvent_t stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);

		accumulate_edge_points << <IMAGE_SIZE, IMAGE_SIZE >> > (dev_image, IMAGE_SIZE, dev_edges_x, dev_edges_y, dev_edges_len);

		// End timer code
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		// Get the elapsed time in milliseconds
		float ms;
		hipEventElapsedTime(&ms, start, stop);
		//printf("Edge array creation: %.3fms\n", ms);
		accumulate_time += ms;

		// Start timer code
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);

		hough << <NUM_RADIUS, 1024 >> > (dev_edges_x, dev_edges_y, dev_edges_len, dev_acc);

		// End timer code
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		// Get the elapsed time in milliseconds
		hipEventElapsedTime(&ms, start, stop);
		//printf("Hough accumulation: %.3fms\n", ms);
		hough_time += ms;

		//Here just for testing.
		hipDeviceSynchronize();

		// Get the accumulator from global memory
		cuda_stat = hipMemcpy(acc, dev_acc, sizeof(int) * TOTAL_ACC_SIZE, hipMemcpyDeviceToHost);
		if (cuda_stat != hipSuccess) {
			printf("Accumulator move to host failed");
			return EXIT_FAILURE;
		}

		ofstream my_file_out((output_dir + "image-" + to_string(z) + "-out"), ios::out | ios::binary);
		if (my_file_out.is_open()) {
			my_file_out.write((char*)acc, sizeof(int) * TOTAL_ACC_SIZE);
			my_file_out.close();
		}
		else {
			cout << "Output file not opened\n";
			return 0;
		}
	}
	hipFree(dev_image);
	hipFree(dev_edges_x);
	hipFree(dev_edges_y);
	hipFree(dev_edges_len);
	hipFree(dev_acc);
	delete[] acc;
	delete[] image;

	printf("Average edge array creation: %.3fms\n", accumulate_time / NUM_IMAGES);
	printf("Average hough accumulation: %.3fms\n", hough_time / NUM_IMAGES);

	return 0;
}